#include "hip/hip_runtime.h"
#define SERVER
#include "utils.h"
#include <hip/hip_runtime.h>
#include <chrono>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16
#define BLOCK_DIM BLOCK_DIM_X * BLOCK_DIM_Y

__global__ void print_array_kernel(uint8_t* array, uint32_t idx) {
    printf("thread : %d, array[%d] = %d\n", blockIdx.x * blockDim.x + threadIdx.x, idx, array[idx]);
    array[idx] = 5;
}

void test_cuda()
{
    // Size of the array
    size_t arraySize = 1000;

    // Host array
    uint8_t* hostArray = new uint8_t[arraySize];
    hostArray[3] = 20;

    // CUDA device array
    uint8_t* deviceArray;

    // Allocate GPU memory
    hipMalloc((void**)&deviceArray, arraySize * sizeof(uint8_t));

    // Copy data from host to GPU
    hipMemcpy(deviceArray, hostArray, arraySize * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Launch the kernel
    print_array_kernel<<<1, 1>>>(deviceArray, 3);
    hipDeviceSynchronize();
    hipMemcpy(hostArray, deviceArray, arraySize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    printf("CPU : array[3] = %d\n", hostArray[3]);


    // Free GPU memory
    hipFree(deviceArray);
}

__global__ void clahe_kernel(uint8_t* in_image, uint8_t* out_image, uint32_t width, uint32_t height, uint32_t clip_limit, uint32_t tileRadius)
{
    uint32_t center_pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t center_pixel_y = blockIdx.y * blockDim.y + threadIdx.y;

    uint32_t mirrored_width = 2 * tileRadius + width,
             mirrored_height = 2 * tileRadius + height;

    float pixel_for_window = (2 * tileRadius + 1) * (2 * tileRadius + 1);
    
    // Check if thread's pixel belongs to the center, if not do nothing
    if(center_pixel_y >= tileRadius && center_pixel_y < mirrored_height - tileRadius && center_pixel_x >= tileRadius && center_pixel_x < mirrored_width - tileRadius)
    {
        // Calculate the histogram for the current tile
        uint32_t histogram[GRAYSCALE_RANGE] = {0};
        uint32_t start_tile_y = center_pixel_y - tileRadius, 
                 end_tile_y = center_pixel_y + tileRadius,
                 start_tile_x = center_pixel_x - tileRadius, 
                 end_tile_x = center_pixel_x + tileRadius;


        // Iterate through pixels in the current tile and calculate histogram
        for(uint32_t tile_pixel_y = start_tile_y; tile_pixel_y < end_tile_y + 1; tile_pixel_y++)
        {
            for(uint32_t tile_pixel_x = start_tile_x; tile_pixel_x < end_tile_x + 1; tile_pixel_x++) 
            {
                uint32_t pixelValue = in_image[tile_pixel_y * mirrored_width + tile_pixel_x];
                histogram[pixelValue]++;
            }
        }

        // Calculate max and min value in histogram to convert clip limit
        uint32_t max_level_value = 0, min_level_value = 1000000;
        for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
        {
            uint32_t level_value = histogram[level];
            if(level_value < min_level_value) min_level_value = level_value;
            if(level_value > max_level_value) max_level_value = level_value;
        }

        uint32_t scaled_clip_limit = min_level_value + (clip_limit * (max_level_value - min_level_value) / 10); 

        // Clip histogram values above the clip limit
        uint32_t excess = 0;
        for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
        {
            if (histogram[level] > scaled_clip_limit)
            {
                excess += histogram[level] - scaled_clip_limit;
                histogram[level] = scaled_clip_limit;
            }
        }

        // Distribute the excess counts uniformly among all histogram bins
        uint32_t tile_cdf[GRAYSCALE_RANGE] = {0}, cdf_counter = 0;
        uint32_t bin_increment = excess / GRAYSCALE_RANGE;
        uint32_t remainder = excess % GRAYSCALE_RANGE;
        for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
        {
            histogram[level] += bin_increment;
        }
        for(uint32_t level = 0; level < remainder; level++)
        {
            histogram[level]++;
        }

        // calculate tile cdf
        uint32_t min_cdf = 0;
        for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
        {
            cdf_counter += histogram[level];
            tile_cdf[level] = cdf_counter;
            if(min_cdf == 0 && cdf_counter != 0) min_cdf = cdf_counter;
        }

        // Apply the contrast-limited histogram equalization to the main pixel
        uint32_t main_pixel_abs_pos = center_pixel_y * mirrored_width + center_pixel_x;
        uint8_t new_pixel_value = std::round(static_cast<double>(tile_cdf[in_image[main_pixel_abs_pos]] - min_cdf) / (pixel_for_window - min_cdf) * (GRAYSCALE_RANGE - 1));
        out_image[(center_pixel_y - tileRadius) * width + (center_pixel_x - tileRadius)] = new_pixel_value;
    }
}

__global__ void clahe_kernel_shared_mem(uint8_t* in_image, uint8_t* out_image, uint32_t width, uint32_t height, uint32_t clip_limit, const uint32_t claheRadius, uint32_t n_pixel_to_load_4_thread)
{
    //_______________________________________________________________________________________________________________
    // FIRST PHASE: load all pixel needed for the current block in shared memory

    extern __shared__ uint8_t tile_shared[];
    uint32_t pixel_block_num = threadIdx.y * blockDim.x + threadIdx.x;
    uint32_t tile_dim_x = (2 * claheRadius + blockDim.x);
    uint32_t tile_dim_y = (2 * claheRadius + blockDim.y);
    uint32_t mirrored_width = 2 * claheRadius + width,
            mirrored_height = 2 * claheRadius + height;
    // calculating range of pixel that current thread need to load 
    uint32_t start_load_index = pixel_block_num * n_pixel_to_load_4_thread;
    uint32_t end_load_index = 0;
    if(pixel_block_num == blockDim.x * blockDim.y - 1)
    {
        //pixels to load are divided equally, last thread load also remaining pixels
        end_load_index = tile_dim_x * tile_dim_y;
    }
    else
    {
        end_load_index = (pixel_block_num +1) * n_pixel_to_load_4_thread;
    }
    
    for(uint32_t pixel_to_load_idx =  start_load_index; pixel_to_load_idx < end_load_index; pixel_to_load_idx++ )
    {
        // mapping pixel_to_load_idx to pixel idx relative to in_image
        uint32_t pixel_idx_y = pixel_to_load_idx / tile_dim_x + blockIdx.y * blockDim.y;
        uint32_t pixel_idx_x = pixel_to_load_idx % tile_dim_x + blockIdx.x * blockDim.x;
        uint32_t abs_pixel_idx = pixel_idx_y * mirrored_width + pixel_idx_x;
        //handle right and bottom blocks (to avoid accessing pixels that don't belong to the image)
        if(pixel_idx_x < mirrored_width && pixel_idx_y < mirrored_height)
        {
            tile_shared[pixel_to_load_idx] = in_image[abs_pixel_idx];
        }
    }

    __syncthreads();
    //_______________________________________________________________________________________________________________
    // SECOND PHASE: calculate clahe

    // avoid pixels out of image
    uint32_t main_pixel_idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    uint32_t main_pixel_idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    if(main_pixel_idx_x < width && main_pixel_idx_y < height)
    {
        // Iterate through pixels in the current tile and calculate histogram
        uint16_t histogram[GRAYSCALE_RANGE] = {0};

        for(uint32_t tile_pixel_y = threadIdx.y; tile_pixel_y < 2 * claheRadius + 1 + threadIdx.y; tile_pixel_y++)
        {
            for(uint32_t tile_pixel_x = threadIdx.x; tile_pixel_x < 2 * claheRadius + 1 + threadIdx.x; tile_pixel_x++) 
            {
                uint8_t pixelValue = tile_shared[tile_pixel_y * tile_dim_x + tile_pixel_x];
                histogram[pixelValue]++;
            }
        }

        //TESTING
        // if(blockIdx.x == 50 && blockIdx.y == 50 && threadIdx.x == 0 && threadIdx.y == 0)
        // {
        //     for(uint32_t pixel_value=0; pixel_value < GRAYSCALE_RANGE; pixel_value++)
        //     {
        //         printf("%d ", histogram[pixel_value]);
        //     }
        //     printf("\n");
        // }

        // Calculate max and min value in histogram to convert clip limit
        uint32_t max_level_value = 0, min_level_value = 1000000;
        for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
        {
            uint32_t level_value = histogram[level];
            if(level_value < min_level_value) min_level_value = level_value;
            if(level_value > max_level_value) max_level_value = level_value;
        }

        uint32_t scaled_clip_limit = min_level_value + (clip_limit * (max_level_value - min_level_value) / 10); 

        // Clip histogram values above the clip limit
        uint32_t excess = 0;
        for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
        {
            if (histogram[level] > scaled_clip_limit)
            {
                excess += histogram[level] - scaled_clip_limit;
                histogram[level] = scaled_clip_limit;
            }
        }

        // Distribute the excess counts uniformly among all histogram bins
        uint32_t cdf_counter = 0;
        uint32_t bin_increment = excess / GRAYSCALE_RANGE;
        uint32_t remainder = excess % GRAYSCALE_RANGE;
        for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
        {
            histogram[level] += bin_increment;
        }
        for(uint32_t level = 0; level < remainder; level++)
        {
            histogram[level]++;
        }

        // calculate window cdf (saving cdf inside histogram array to reduce local memory)
        uint32_t min_cdf = 0;
        for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
        {
            cdf_counter += histogram[level];
            histogram[level] = cdf_counter;
            if(min_cdf == 0 && cdf_counter != 0) min_cdf = cdf_counter;
        }

        //TESTING
        // if(blockIdx.x == 50 && blockIdx.y == 50 && threadIdx.x == 0 && threadIdx.y == 0)
        // {
        //     for(uint32_t pixel_value=0; pixel_value < GRAYSCALE_RANGE; pixel_value++)
        //     {
        //         printf("%d ", window_cdf[pixel_value]);
        //     }
        //     printf("\n");
        // }

        // Apply the contrast-limited histogram equalization to the main pixel

        float pixel_for_window = (2 * claheRadius + 1) * (2 * claheRadius + 1);
        uint32_t main_pixel_abs_pos = main_pixel_idx_y * width + main_pixel_idx_x;
        uint32_t mirrored_pixel_abs_pos = (main_pixel_idx_y + claheRadius) * mirrored_width + main_pixel_idx_x + claheRadius;
        uint8_t new_pixel_value = std::round(static_cast<double>(histogram[in_image[mirrored_pixel_abs_pos]] - min_cdf) / (pixel_for_window - min_cdf) * (GRAYSCALE_RANGE - 1));
        out_image[main_pixel_abs_pos] = new_pixel_value;

        //TESTING
        // if(blockIdx.x == 70 && blockIdx.y == 50 && threadIdx.x == 0 && threadIdx.y == 0)
        // {
        //     printf("pixel pos : %d, cdf : %d, new_pixel_value : %d, min_cdf : %d\n", mirrored_pixel_abs_pos, window_cdf[in_image[mirrored_pixel_abs_pos]], new_pixel_value, min_cdf);
        // }
    }
}

void parallel_clahe(uint32_t clip_limit, uint32_t tileRadius, std::string in_img_path , std::string out_img_path)
{
    int width, height, channels;
    uint8_t *input_img = stbi_load(in_img_path.c_str(), &width, &height, &channels, 0);
    if(input_img == NULL) {
        std::cout << "Error loading image\n";
    }
    else
    {
        // std::cout << "width: " <<width << " height: " << height << " channels: " << channels << "\n";
    }

    uint8_t *output_img = (uint8_t* )malloc(width * height * sizeof(uint8_t));
    memset(output_img, 0, width * height * sizeof(uint8_t));

    //Border mirroring
    uint32_t mirrored_width = 2 * tileRadius + width,
             mirrored_height = 2 * tileRadius + height;

    uint8_t* in_mirrored_image = (uint8_t*)malloc((mirrored_height) * (mirrored_width) * sizeof(uint8_t)); 
    memset(in_mirrored_image, 0, (mirrored_height) * (mirrored_width) * sizeof(uint8_t));

    mirror_img_borders(in_mirrored_image, input_img, mirrored_height, mirrored_width, height, width, tileRadius);

    // CUDA device imgs
    uint8_t* gpu_input_img;
    uint8_t* gpu_output_img;

    // Allocate GPU memory
    hipMalloc((void**)&gpu_input_img, (mirrored_height) * (mirrored_width) * sizeof(uint8_t));
    hipMalloc((void**)&gpu_output_img, width * height * sizeof(uint8_t));

    // Copy data from host to GPU
    hipMemcpy(gpu_input_img, in_mirrored_image, (mirrored_height) * (mirrored_width) * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Launch the kernel
    dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridDim((mirrored_width + blockDim.x - 1) / blockDim.x, (mirrored_height + blockDim.y - 1) / blockDim.y);

    clahe_kernel<<<gridDim, blockDim>>>(gpu_input_img, gpu_output_img, width, height, clip_limit, tileRadius);
    hipDeviceSynchronize();
    
    hipMemcpy(output_img, gpu_output_img, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(gpu_input_img);
    hipFree(gpu_output_img);

    stbi_write_png(out_img_path.c_str(), width, height, 1, output_img, width);
}

void parallel_clahe_shared_mem(uint32_t clip_limit, uint32_t tileRadius, std::string in_img_path , std::string out_img_path)
{
    int width, height, channels;
    uint8_t *input_img = stbi_load(in_img_path.c_str(), &width, &height, &channels, 0);
    if(input_img == NULL) {
        std::cout << "Error loading image\n";
    }
    else
    {
        // std::cout << "width: " <<width << " height: " << height << " channels: " << channels << "\n";
    }

    uint8_t *output_img = (uint8_t* )malloc(width * height * sizeof(uint8_t));
    memset(output_img, 0, width * height * sizeof(uint8_t));

    //Border mirroring
    uint32_t mirrored_width = 2 * tileRadius + width,
             mirrored_height = 2 * tileRadius + height;

    uint8_t* in_mirrored_image = (uint8_t*)malloc((mirrored_height) * (mirrored_width) * sizeof(uint8_t)); 
    memset(in_mirrored_image, 0, (mirrored_height) * (mirrored_width) * sizeof(uint8_t));

    mirror_img_borders(in_mirrored_image, input_img, mirrored_height, mirrored_width, height, width, tileRadius);

    // CUDA device imgs
    uint8_t* gpu_input_img;
    uint8_t* gpu_output_img; 

    // Allocate GPU memory
    hipMalloc((void**)&gpu_input_img, (mirrored_height) * (mirrored_width) * sizeof(uint8_t));
    hipMalloc((void**)&gpu_output_img, width * height * sizeof(uint8_t));

    // Copy data from host to GPU
    hipMemcpy(gpu_input_img, in_mirrored_image, (mirrored_height) * (mirrored_width) * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Launch the kernel
    dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    uint32_t n_pixel_to_load_4_thread = ((2 * tileRadius + BLOCK_DIM_X) * (2 * tileRadius + BLOCK_DIM_Y ))/ (BLOCK_DIM_X * BLOCK_DIM_Y); 

    // TESTING_________________________________
    // for(uint32_t pixel_idx_y = 4; pixel_idx_y < 14; pixel_idx_y++)
    // {
    //     for(uint32_t pixel_idx_x = 0; pixel_idx_x < 10; pixel_idx_x ++)
    //     {
    //         printf("%d ", in_mirrored_image[pixel_idx_y * mirrored_width + pixel_idx_x]);
    //     }
    //     printf("\n");
    // }
    // printf("\n");
    //__________________________________________

    clahe_kernel_shared_mem<<<gridDim, blockDim, (2 * tileRadius + BLOCK_DIM_X) * (2 * tileRadius + BLOCK_DIM_Y ) * sizeof(uint8_t) >>>(gpu_input_img, gpu_output_img, width, height, clip_limit, tileRadius, n_pixel_to_load_4_thread);
    hipDeviceSynchronize();
    
    hipMemcpy(output_img, gpu_output_img, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(gpu_input_img);
    hipFree(gpu_output_img);

    stbi_write_png(out_img_path.c_str(), width, height, 1, output_img, width);
}

void cpu_clahe(uint8_t* in_image, uint8_t* out_image, uint32_t width, uint32_t height, uint32_t clip_limit, uint32_t tileRadius)
{
    uint32_t mirrored_width = 2 * tileRadius + width,
             mirrored_height = 2 * tileRadius + height;
    
    float pixel_for_window = (2 * tileRadius + 1) * (2 * tileRadius + 1);
    // Iterate through each pixel (need offset because of the mirrored border)
    for(uint32_t main_pixel_y = tileRadius; main_pixel_y < mirrored_height - tileRadius; main_pixel_y++)
    {
        for(uint32_t main_pixel_x = tileRadius; main_pixel_x < mirrored_width - tileRadius; main_pixel_x++)
        {
            // Calculate the histogram for the current tile
            uint32_t histogram[GRAYSCALE_RANGE] = {0};
            uint32_t start_tile_y = main_pixel_y - tileRadius, 
                     end_tile_y = main_pixel_y + tileRadius,
                     start_tile_x = main_pixel_x - tileRadius, 
                     end_tile_x = main_pixel_x + tileRadius;


            // Iterate through pixels in the current tile and calculate histogram
            for(uint32_t tile_pixel_y = start_tile_y; tile_pixel_y < end_tile_y + 1; tile_pixel_y++)
            {
                for(uint32_t tile_pixel_x = start_tile_x; tile_pixel_x < end_tile_x + 1; tile_pixel_x++) 
                {
                    uint32_t pixelValue = in_image[tile_pixel_y * mirrored_width + tile_pixel_x];
                    histogram[pixelValue]++;
                }
            }

            // TESTING
            // if(main_pixel_y == tileRadius + 150 && main_pixel_x == tileRadius + 150)
            // {
            //     for(uint32_t pixel_value=0; pixel_value < GRAYSCALE_RANGE; pixel_value++)
            //     {
            //         printf("%d ", histogram[pixel_value]);
            //     }
            //     printf("\n");
            // }
            // plotHist(histogram);

            // Calculate max and min value in histogram to convert clip limit
            uint32_t max_level_value = 0, min_level_value = 1000000;
            for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
            {
                uint32_t level_value = histogram[level];
                if(level_value < min_level_value) min_level_value = level_value;
                if(level_value > max_level_value) max_level_value = level_value;
            }

            uint32_t scaled_clip_limit = min_level_value + (clip_limit * (max_level_value - min_level_value) / 10); 

            // Clip histogram values above the clip limit
            uint32_t excess = 0;
            for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
            {
                if (histogram[level] > scaled_clip_limit)
                {
                    excess += histogram[level] - scaled_clip_limit;
                    histogram[level] = scaled_clip_limit;
                }
            }

            // Distribute the excess counts uniformly among all histogram bins
            uint32_t tile_cdf[GRAYSCALE_RANGE] = {0}, cdf_counter = 0;
            uint32_t bin_increment = excess / GRAYSCALE_RANGE;
            uint32_t remainder = excess % GRAYSCALE_RANGE;
            for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
            {
                histogram[level] += bin_increment;
            }
            for(uint32_t level = 0; level < remainder; level++)
            {
                histogram[level]++;
            }

            // calculate tile cdf
            uint32_t min_cdf = 0;
            for(uint32_t level = 0; level < GRAYSCALE_RANGE; level++)
            {
                cdf_counter += histogram[level];
                tile_cdf[level] = cdf_counter;
                if(min_cdf == 0 && cdf_counter != 0) min_cdf = cdf_counter;
            }

            // TESTING
            // if(main_pixel_y == tileRadius + 150 && main_pixel_x == tileRadius + 150)
            // {
            //     for(uint32_t pixel_value=0; pixel_value < GRAYSCALE_RANGE; pixel_value++)
            //     {
            //         printf("%d ", tile_cdf[pixel_value]);
            //     }
            //     printf("\n");
            // }
            // plotHist(tile_cdf);

            // Apply the contrast-limited histogram equalization to the main pixel
            uint32_t main_pixel_abs_pos = main_pixel_y * mirrored_width + main_pixel_x;
            uint8_t new_pixel_value = std::round(static_cast<double>(tile_cdf[in_image[main_pixel_abs_pos]] - min_cdf) / (pixel_for_window - min_cdf) * (GRAYSCALE_RANGE - 1));
            out_image[(main_pixel_y - tileRadius) * width + (main_pixel_x - tileRadius)] = new_pixel_value;

            // TESTING
            // if(main_pixel_y == tileRadius + 150 && main_pixel_x == tileRadius + 210)
            // {
            //     printf("pixel pos : %d, cdf : %d, new_pixel_value : %d, min_cdf : %d\n", main_pixel_abs_pos, tile_cdf[in_image[main_pixel_abs_pos]], new_pixel_value, min_cdf);
            // }
        }
    }
}

void single_core_clahe(uint32_t clip_limit, uint32_t tileRadius)
{
    std::string in_img_path = "./../media/test3.jpg";
    std::string out_img_path = "./../media/test3_output_cpu.jpg";

    int width, height, channels;
    uint8_t *input_img = stbi_load(in_img_path.c_str(), &width, &height, &channels, 0);
    if(input_img == NULL) {
        std::cout << "Error loading image\n";
    }
    else
    {
        std::cout << "width: " <<width << " height: " << height << " channels: " << channels << "\n";
    }

    uint8_t *output_img = (uint8_t* )malloc(width * height * sizeof(uint8_t));
    memset(output_img, 0, width * height * sizeof(uint8_t));

    //Border mirroring
    uint32_t mirrored_width = 2 * tileRadius + width,
             mirrored_height = 2 * tileRadius + height;

    uint8_t* in_mirrored_image = (uint8_t*)malloc((mirrored_height) * (mirrored_width) * sizeof(uint8_t)); 
    memset(in_mirrored_image, 0, (mirrored_height) * (mirrored_width) * sizeof(uint8_t));

    mirror_img_borders(in_mirrored_image, input_img, mirrored_height, mirrored_width, height, width, tileRadius);

    cpu_clahe(in_mirrored_image, output_img, width, height, clip_limit, tileRadius);

    stbi_write_png(out_img_path.c_str(), width, height, 1, output_img, width);
}

void benchmark()
{
    uint32_t iter_n = 100;

    auto start_time = std::chrono::high_resolution_clock::now();
    for(uint32_t iter_idx=0; iter_idx<iter_n; iter_idx++)
    {
        if(iter_idx == iter_n - 1)
        {
            std::cout << "Iteration : ["<< iter_idx + 1 << "/" << iter_n << "]\n";
        }
        else
        {
            std::cout << "Iteration : ["<< iter_idx + 1 << "/" << iter_n << "]" << std::flush << "\r";
        }
        // parallel_clahe(4, 40, "test3.jpg");
    }
    auto end_time = std::chrono::high_resolution_clock::now();
    auto elapsed_time = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time);
    std::cout << "Gpu version time elapsed : " << elapsed_time.count() << "\n";

    start_time = std::chrono::high_resolution_clock::now();
    for(uint32_t iter_idx=0; iter_idx<iter_n; iter_idx++)
    {
        if(iter_idx == iter_n - 1)
        {
            std::cout << "Iteration : ["<< iter_idx + 1 << "/" << iter_n << "]\n";
        }
        else
        {
            std::cout << "Iteration : ["<< iter_idx + 1 << "/" << iter_n << "]" << std::flush << "\r";
        }
        // parallel_clahe_shared_mem(4, 40, "test3.jpg");
    }
    end_time = std::chrono::high_resolution_clock::now();
    elapsed_time = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time);
    std::cout << "Gpu shared memory version time elapsed : " << elapsed_time.count() << "\n";


}

int main()
{
    // CPU
    auto start_time = std::chrono::high_resolution_clock::now();
    // single_core_clahe(4, 20);
    auto end_time = std::chrono::high_resolution_clock::now();
    auto elapsed_time = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time);
    // std::cout << "Cpu version time elapsed : " << elapsed_time.count() << "\n";

    uint32_t img_count = std::distance(fs::directory_iterator("./../media/input/grayscale_images/"), fs::directory_iterator{});
    uint32_t benchmark_limit = 1000;

    // GPU
    start_time = std::chrono::high_resolution_clock::now();
    uint32_t iter_counter = 0;
    for (const auto& entry : fs::directory_iterator("./../media/input/grayscale_images/")) {
            std::string img_name =  entry.path().filename().string();
            std::string in_img_path = "./../media/input/grayscale_images/" + img_name;
            std::string out_img_path = "./../media/output/gpu_" + img_name;
            parallel_clahe(4, 40, in_img_path, out_img_path);
            if(iter_counter == img_count - 1)
            {
                std::cout << "Iteration : ["<< iter_counter + 1 << "/" << img_count << "]\n";
            }
            else
            {
                std::cout << "Iteration : ["<< iter_counter + 1 << "/" << img_count << "]" << std::flush << "\r";
            }
            iter_counter++;
            if(iter_counter >= benchmark_limit)break;
    }
    end_time = std::chrono::high_resolution_clock::now();
    elapsed_time = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time);
    std::cout << "Gpu version time elapsed : " << elapsed_time.count() << "\n";

    //GPU shared memory
    start_time = std::chrono::high_resolution_clock::now();
    iter_counter = 0;
    for (const auto& entry : fs::directory_iterator("./../media/input/grayscale_images/")) {
            std::string img_name =  entry.path().filename().string();
            std::string in_img_path = "./../media/input/grayscale_images/" + img_name;
            std::string out_img_path = "./../media/output/gpu_shared_mem_" + img_name;
            parallel_clahe_shared_mem(4, 40, in_img_path, out_img_path);
            if(iter_counter == img_count - 1)
            {
                std::cout << "Iteration : ["<< iter_counter + 1 << "/" << img_count << "]\n";
            }
            else
            {
                std::cout << "Iteration : ["<< iter_counter + 1 << "/" << img_count << "]" << std::flush << "\r";
            }
            iter_counter++;
            if(iter_counter >= benchmark_limit)break;
    }
    end_time = std::chrono::high_resolution_clock::now();
    elapsed_time = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time);
    std::cout << "Gpu shared memory version time elapsed : " << elapsed_time.count() << "\n";
    
    // benchmark();

    // start_time = std::chrono::high_resolution_clock::now();
    // parallel_clahe_shared_mem(4,20, "./../media/test_img/test3.jpg", "./../media/test_img/test3_output_cuda_shared.jpg");
    // end_time = std::chrono::high_resolution_clock::now();
    // elapsed_time = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time);
    // std::cout << "Gpu shared memory version time elapsed : " << elapsed_time.count() << "\n";
}